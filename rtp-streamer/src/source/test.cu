#include "hip/hip_runtime.h"
#include "test.cuh"

__global__ void test_kernel(void) {
	
}

__global__ void VecAddKernel(float* A, float* B, float* C, int N) {
	int i = threadIdx.x;
	if (i < N) {
		C[i] = A[i] + B[i];
	}
}

namespace Wrapper {
void wrapper(void) {
	test_kernel <<<1, 1>>> ();
	printf("Hello CUDA");
}
void runVecAdd(float* A, float* B, float* C, int N) {
	float* d_A, *d_B, *d_C;

	hipMalloc((void**)&d_A, N * sizeof(float));
	hipMalloc((void**)&d_B, N * sizeof(float));
	hipMalloc((void**)&d_C, N * sizeof(float));

	hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);

	// Launch the kernel
	VecAddKernel<<<1, N>>>(d_A, d_B, d_C, N);

	hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}
}